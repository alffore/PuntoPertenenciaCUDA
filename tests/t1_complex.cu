#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <string>


int main(){

hipDoubleComplex test,otro,res;

    test.x =0;
    test.y=1;

    otro.x=4;
    otro.y=M_PI;

    res=hipCmul(test,otro);

    std::cout<<res.x << " "<<res.y<<std::endl;

    std::cout<< sizeof(hipDoubleComplex) << " "<<sizeof(double)<< std::endl;

    double t=0.001;

    std::cout<< hipCmul(make_hipDoubleComplex(t,0),otro).x <<" "<< hipCmul(make_hipDoubleComplex(t,0),otro).y << std::endl;

    return 0;
}