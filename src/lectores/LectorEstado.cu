#include "../PuntoPertenecia.h"
#include "../PuntoPertenciaCUDA.h"

long total_vertices_pest;

extern void split(vector<string> &theStringVector, const string &theString, const string &theDelimiter);

int cargaArchivoEstado(string sarchivo, string sep, PEstado pest);
void parserEstado(string scad, string sep, PEstado pest);

/*bool checacuCDC(hipDoubleComplex z1,hipDoubleComplex z2);
void parserEstado_v2(string scad,string sep,PEstado pest);*/

/**
 * @brief
 *
 * @param sarchivo
 * @param sep
 * @param pest
 * @return int
 */
int cargaArchivoEstado(string sarchivo, string sep, PEstado pest)
{

    string sline;

    ifstream miarch;

    miarch.open(sarchivo.c_str(), ifstream::in);

    unsigned int i = 0;
    while (getline(miarch, sline))
    {
        parserEstado(sline, sep, pest + i);
        i++;
    }

    miarch.close();

    return 0;
}

/**
 * @brief
 *
 * @param scad
 * @param sep
 */
void parserEstado(string scad, string sep, PEstado pest)
{

    vector<string> vc;

    split(vc, scad, sep);
    // std::cout<<vc[0]<<" "<<vc[1]<<std::endl;

    pest->nvertices = atoi(vc[1].c_str());
    total_vertices_pest += pest->nvertices;

    pest->x = (float *)malloc(sizeof(float) * pest->nvertices);
    pest->y = (float *)malloc(sizeof(float) * pest->nvertices);

    pest->id = atoi(vc[3].c_str());

    vector<string> vsc;
    split(vsc, vc[2], ",");

    cout << pest->nvertices << "::" << vsc.size() << endl;

    for (unsigned int j = 0; j < pest->nvertices; j++)
    {
        *(pest->x + j) = atof(vsc[2 * j].c_str()) / 10000;
        *(pest->y + j) = atof(vsc[2 * j + 1].c_str()) / 10000;
    }
}

/*
void parserEstado_v2(string scad,string sep,PEstado pest) {

    vector<string> vc;

    split(vc, scad, sep);

    unsigned int vertices = atoi(vc[1].c_str());

    pest->id=atoi(vc[3].c_str());

    vector<string> vsc;
    split(vsc, vc[2], ",");

    vector<hipDoubleComplex> v_c;


    for(unsigned int j =0; j<vertices; j++){

        v_c.push_back(make_hipDoubleComplex(atof(vsc[2*j].c_str())/1000,
        atof(vsc[2*j+1].c_str())/1000));

    }

    cout<<"e: " << pest->id<<" vector size: "<<v_c.size();

    v_c.erase(unique(v_c.begin(),v_c.end(),checacuCDC),v_c.end());

    cout<<" vector size unique erase: "<<v_c.size()<<endl;
}


bool checacuCDC(hipDoubleComplex z1,hipDoubleComplex z2){
    return (z1.x==z2.x && z1.y==z2.y);
}
*/