#include "../PuntoPertenecia.h"
#include "../PuntoPertenciaCUDA.h"

void alojaMemMNZ(vector<Manzana> &vmnz, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);
void liberaMemMNZ(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);

/**
 * @brief
 *
 * @param vagb
 * @param host_dep
 * @param device_dep
 * @param host_drefp
 * @param device_drefp
 */
void alojaMemMNZ(vector<Manzana> &vmnz, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp)
{

    size_t tam_pol = vmnz.size();
    size_t tam_coord = 0;

    host_drefp = (PDRefP)malloc(tam_pol * sizeof(DRefP));

    for (auto &e : vmnz)
    {
        tam_coord += e.vp.size();
    }

    host_dep = (PDEP)malloc(tam_coord * sizeof(DEP));

    size_t pos = 0;
    for (size_t i = 0; i < tam_pol; i++)
    {
        (host_drefp + i)->ini = pos;

        size_t tam_vp = vmnz[i].vp.size();
        for (size_t j = 0; j < tam_vp; j++)
        {
            (host_dep + pos + j)->x = vmnz[i].vp[j].x;
            (host_dep + pos + j)->y = vmnz[i].vp[j].y;
        }

        pos += vmnz[i].vp.size();
        (host_drefp + i)->fin = pos;
    }

    hipMalloc((void **)&device_dep, tam_coord * sizeof(DEP));
    hipMemcpy(device_dep, host_dep, tam_coord * sizeof(DEP), hipMemcpyHostToDevice);

    hipMalloc((void **)&device_drefp, tam_pol * sizeof(DRefP));
    hipMemcpy(device_drefp, host_drefp, tam_pol * sizeof(DRefP), hipMemcpyHostToDevice);
}

/**
 * @brief
 *
 * @param host_dep
 * @param device_dep
 * @param host_drefp
 * @param device_drefp
 */
void liberaMemMNZ(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp)
{
    hipFree(device_dep);
    hipFree(device_drefp);

    free(host_dep);
    free(host_drefp);
}