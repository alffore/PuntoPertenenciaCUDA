#include "hip/hip_runtime.h"
#include "../PuntoPertenciaCUDA.h"

/**
 * @brief
 *
 * @param d_dep
 * @param d_prec
 * @param ini
 * @param fin
 * @return __device__
 */
__device__ float sumaAngulos(PDEP d_dep, PDRec d_prec, size_t ini, size_t fin)
{
    float angulo = 0.0;

    float x1 = (d_dep + ini)->x - d_prec->x;
    float y1 = (d_dep + ini)->y - d_prec->y;

    float x0 = (d_dep + fin - 1)->x - d_prec->x;
    float y0 = (d_dep + fin - 1)->y - d_prec->y;

    float mag = sqrt(x0 * x0 + y0 * y0) * sqrt(x1 * x1 + y1 * y1);

    float aux = (x0 * y1 - x1 * y0) / mag;

    if (aux > 1.0)
        aux = 1.0;
    if (aux < -1.0)
        aux = -1.0;

    angulo += asinf(aux);

    for (size_t i = ini + 1; i < fin; i++)
    {
        x1 = (d_dep + i)->x - d_prec->x;
        y1 = (d_dep + i)->y - d_prec->y;

        x0 = (d_dep + i - 1)->x - d_prec->x;
        y0 = (d_dep + i - 1)->y - d_prec->y;

        mag = sqrt(x0 * x0 + y0 * y0) * sqrt(x1 * x1 + y1 * y1);

        /*if (mag == 0.0)
        {
            aux = 0.0;
        }
        else
        {*/
            aux = (x0 * y1 - x1 * y0) / mag;
       // }

        if (aux > 1.0)
            aux = 1.0;
        if (aux < -1.0)
            aux = -1.0;

        angulo += asinf(aux);
    }

    return angulo;
}

/**
 * @brief
 *
 * @param device_dep
 * @param device_drefp
 * @param device_pdrec
 * @param num_rec
 * @param num_pol
 * @return __global__
 */
__global__ void kernel_polpertenciaEstado(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t num_pol)
{
    unsigned int gtid = threadIdx.x + blockIdx.x * blockDim.x;

    if (gtid < num_rec)
    {
        for (size_t i = 0; i < num_pol; i++)
        {
            float angulo = fabsf(sumaAngulos(device_dep, (device_pdrec + gtid), (device_drefp + i)->ini, (device_drefp + i)->fin));
            if (angulo >= 6.28)
            {
                (device_pdrec + gtid)->id_e = i;
                (device_pdrec + gtid)->res = angulo;
            }
        }
    }
}

/**
 * @brief
 *
 * @param device_dep
 * @param device_drefp
 * @param device_pdrec
 * @param num_rec
 * @param num_pol
 * @return __global__
 */
__global__ void kernel_polpertenciaMunicipio(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t num_pol)
{
    unsigned int gtid = threadIdx.x + blockIdx.x * blockDim.x;

    if (gtid < num_rec)
    {
        for (size_t i = 0; i < num_pol; i++)
        {
            float angulo = fabsf(sumaAngulos(device_dep, (device_pdrec + gtid), (device_drefp + i)->ini, (device_drefp + i)->fin));
            if (angulo >= 6.28)
            {
                (device_pdrec + gtid)->id_m = i;
                (device_pdrec + gtid)->res = angulo;
            }
        }
    }
}

/**
 * @brief 
 * 
 * @param device_dep 
 * @param device_drefp 
 * @param device_pdrec 
 * @param num_rec 
 * @param ini_pol 
 * @param fin_pol 
 * @return __global__ 
 */
__global__ void kernel_polpertenciaAGEB(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t ini_pol, size_t fin_pol)
{
    unsigned int gtid = threadIdx.x + blockIdx.x * blockDim.x;

    if (gtid < num_rec)
    {
        //(device_pdrec + gtid)->res = 0;
        for (size_t i = ini_pol; i < fin_pol; i++)
        {
            float angulo = fabsf(sumaAngulos(device_dep, (device_pdrec + gtid), (device_drefp + i)->ini, (device_drefp + i)->fin));
            if (angulo >= 6.28)
            {
                (device_pdrec + gtid)->id_ageb = i;
                (device_pdrec + gtid)->res = angulo;
            }
        }
    }
}

__global__ void kernel_polpertenciaMNZ(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t ini_pol, size_t fin_pol)
{
    unsigned int gtid = threadIdx.x + blockIdx.x * blockDim.x;

    if (gtid < num_rec)
    {
        if((device_pdrec+gtid)->id_ageb<0){
            return;
        }
        //(device_pdrec + gtid)->res = 0;
        for (size_t i = ini_pol; i < fin_pol; i++)
        {
            float angulo = fabsf(sumaAngulos(device_dep, (device_pdrec + gtid), (device_drefp + i)->ini, (device_drefp + i)->fin));
            if (angulo >= 6.28)
            {
                (device_pdrec + gtid)->id_mnz= i;
                (device_pdrec + gtid)->res = angulo;
            }
        }
    }
}