#include "hip/hip_runtime.h"
#include "../PuntoPertenciaCUDA.h"

/**
 * @brief
 *
 * @param d_dep
 * @param d_prec
 * @param ini
 * @param fin
 * @return __device__
 */
__device__ float sumaAngulos(PDEP d_dep, PDRec d_prec, size_t ini, size_t fin)
{
    float angulo = 0.0;

    float x1 = (d_dep + ini)->x - d_prec->x;
    float y1 = (d_dep + ini)->y - d_prec->y;

    float x0 = (d_dep + fin - 1)->x - d_prec->x;
    float y0 = (d_dep + fin - 1)->y - d_prec->y;

    float mag = sqrt(x0 * x0 + y0 * y0) * sqrt(x1 * x1 + y1 * y1);

    float aux = (x0 * y1 - x1 * y0) / mag;

    if (aux > 1.0)
        aux = 1.0;
    if (aux < -1.0)
        aux = -1.0;

    angulo += asinf(aux);

    for (size_t i = ini + 1; i < fin; i++)
    {
        x1 = (d_dep + i)->x - d_prec->x;
        y1 = (d_dep + i)->y - d_prec->y;

        x0 = (d_dep + i - 1)->x - d_prec->x;
        y0 = (d_dep + i - 1)->y - d_prec->y;

        mag = sqrt(x0 * x0 + y0 * y0) * sqrt(x1 * x1 + y1 * y1);

        aux = (x0 * y1 - x1 * y0) / mag;

        if (aux > 1.0)
            aux = 1.0;
        if (aux < -1.0)
            aux = -1.0;
            
        angulo += asinf(aux);
    }

    return angulo;
}

/**
 * @brief
 *
 * @param device_dep
 * @param device_drefp
 * @param device_pdrec
 * @param num_rec
 * @param num_pol
 * @return __global__
 */
__global__ void kernel_polpertenciaEstado(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t num_pol)
{
    unsigned int gtid = threadIdx.x + blockIdx.x * blockDim.x;

    if (gtid < num_rec)
    {
        for (size_t i = 0; i < num_pol; i++)
        {
            float angulo = fabsf(sumaAngulos(device_dep, (device_pdrec + gtid), (device_drefp + i)->ini, (device_drefp + i)->fin));
            if (angulo >= 6.28)
            {
                (device_pdrec + gtid)->id_e = i;
                (device_pdrec + gtid)->res = angulo;
            }
        }
    }
}