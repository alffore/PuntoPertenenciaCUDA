#include "../PuntoPertenecia.h"
#include "../PuntoPertenciaCUDA.h"

void alojaMemRecurso(vector<Recurso> &vrec, PDRec &host_pdrec, PDRec &device_pdrec);
void liberaMemRecurso(PDRec &host_pdrec, PDRec &device_pdrec);

/**
 * @brief
 *
 * @param vrec
 * @param host_pdrec
 * @param device_pdrec
 */
void alojaMemRecurso(vector<Recurso> &vrec, PDRec &host_pdrec, PDRec &device_pdrec)
{
    size_t tam = vrec.size();

    host_pdrec = (PDRec)malloc(tam * sizeof(DRec));

    for (size_t i = 0; i < tam; i++)
    {
        (host_pdrec + i)->id_e = -1;
        (host_pdrec + i)->id_m = -1;
        (host_pdrec + i)->id_l = -1;

        (host_pdrec + i)->id_ageb = -1;
        (host_pdrec + i)->id_mnz = -1;

        (host_pdrec + i)->x = vrec[i].p.x;
        (host_pdrec + i)->y = vrec[i].p.y;
    }

    hipMalloc((void **)&device_pdrec, tam * sizeof(DRec));
    hipMemcpy(device_pdrec, host_pdrec, tam * sizeof(DRec), hipMemcpyHostToDevice);
}

/**
 * @brief
 *
 * @param host_pdrec
 * @param device_pdrec
 */
void liberaMemRecurso(PDRec &host_pdrec, PDRec &device_pdrec)
{
    hipFree(device_pdrec);
    free(host_pdrec);
}