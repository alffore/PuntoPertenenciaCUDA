#include "hip/hip_runtime.h"
#include "../PuntoPertenciaCUDA.h"
#include "../PuntoPertenecia.h"

extern vector<Recurso> vrec;

extern vector<Estado> vest;
extern vector<Municipio> vmun;
extern vector<Ageb> vageb;
extern vector<Manzana> vmnz;

PDRec host_pdrec = nullptr;
PDRec device_pdrec = nullptr;

PDEP host_dep = nullptr;
PDEP device_dep = nullptr;

PDRefP host_drefp = nullptr;
PDRefP device_drefp = nullptr;

void algoritmo();
void marcaRecursos();

// Aloja memoria
extern void alojaMemRecurso(vector<Recurso> &vrec, PDRec &host_pdrec, PDRec &device_pdrec);
extern void liberaMemRecurso(PDRec &host_pdrec, PDRec &device_pdrec);

extern void alojaMemEstado(vector<Estado> &vest, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);
extern void liberaMemEstado(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);

extern void alojaMemMunicipio(vector<Municipio> &vmun, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);
extern void liberaMemMunicipio(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);

extern void alojaMemAGEB(vector<Ageb> &vageb, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);
extern void liberaMemAGEB(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);

extern void alojaMemMNZ(vector<Manzana> &vmnz, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);
extern void liberaMemMNZ(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);

// kernel
extern __global__ void kernel_polpertenciaEstado(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t num_pol);
extern __global__ void kernel_polpertenciaMunicipio(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t num_pol);
extern __global__ void kernel_polpertenciaAGEB(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t ini_pol, size_t fin_pol);
extern __global__ void kernel_polpertenciaMNZ(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t ini_pol, size_t fin_pol);

/**
 * @brief
 *
 */
void algoritmo()
{
    hipSetDevice(1);
    size_t tam_rec = vrec.size();

    int canti_hilos = 1024;
    int canti_bloques = (int)ceil((float)tam_rec / canti_hilos) + 1;

    alojaMemRecurso(vrec, host_pdrec, device_pdrec);

    // kernel estado
    alojaMemEstado(vest, host_dep, device_dep, host_drefp, device_drefp);
    kernel_polpertenciaEstado<<<canti_bloques, canti_hilos>>>(device_dep, device_drefp, device_pdrec, tam_rec, vest.size());
    liberaMemEstado(host_dep, device_dep, host_drefp, device_drefp);

    // kernel municipio
    alojaMemMunicipio(vmun, host_dep, device_dep, host_drefp, device_drefp);
    kernel_polpertenciaMunicipio<<<canti_bloques, canti_hilos>>>(device_dep, device_drefp, device_pdrec, tam_rec, vmun.size());
    liberaMemMunicipio(host_dep, device_dep, host_drefp, device_drefp);

    //kernel ageb
    size_t corte=(size_t)vageb.size()/2;
    alojaMemAGEB(vageb, host_dep,device_dep, host_drefp, device_drefp);
    kernel_polpertenciaAGEB<<<canti_bloques, canti_hilos>>>(device_dep, device_drefp, device_pdrec, tam_rec, 0,corte);
    liberaMemAGEB(host_dep, device_dep, host_drefp, device_drefp);
   
    alojaMemAGEB(vageb, host_dep,device_dep, host_drefp, device_drefp);
    kernel_polpertenciaAGEB<<<canti_bloques, canti_hilos>>>(device_dep, device_drefp, device_pdrec, tam_rec, corte,vageb.size());
    liberaMemAGEB(host_dep, device_dep, host_drefp, device_drefp);

    //kernel Manzanas
    alojaMemMNZ(vmnz, host_dep,device_dep, host_drefp, device_drefp);
    kernel_polpertenciaMNZ<<<canti_bloques, canti_hilos>>>(device_dep, device_drefp, device_pdrec, tam_rec, 0,/*50000*/vmnz.size());
    liberaMemMNZ(host_dep, device_dep, host_drefp, device_drefp);

    // recuperamos la memoria de los recursos
    hipMemcpy(host_pdrec, device_pdrec, tam_rec * sizeof(DRec), hipMemcpyDeviceToHost);

    marcaRecursos();

    liberaMemRecurso(host_pdrec, device_pdrec);
}

/**
 * @brief
 *
 */
void marcaRecursos()
{

    size_t tam = vrec.size();

    for (size_t i = 0; i < tam; i++)
    {
        long id_e = (host_pdrec + i)->id_e;
        if (id_e >= 0)
        {
            vrec[i].ne = vest[id_e].id;
            vrec[i].res = (host_pdrec + i)->res;
        }

        long id_m = (host_pdrec + i)->id_m;
        if (id_m >= 0)
        {
            vrec[i].nm = vmun[id_m].id -vrec[i].ne*1000;
            vrec[i].res = (host_pdrec + i)->res;
        }

        long id_ageb = (host_pdrec + i)->id_ageb;
        if(id_ageb >=0){
            vrec[i].ne= vageb[id_ageb].e;
            vrec[i].nm= vageb[id_ageb].m;
            vrec[i].nl= vageb[id_ageb].l;
            vrec[i].sid_ageb = vageb[id_ageb].sid;
            vrec[i].res = (host_pdrec + i)->res;
        }

        long id_mnz =(host_pdrec + i)->id_mnz;
        if(id_mnz>=0){
            vrec[i].ne= vmnz[id_mnz].e;
            vrec[i].nm= vmnz[id_mnz].m;
            vrec[i].nl= vmnz[id_mnz].l;
            vrec[i].sid_mnz = vmnz[id_mnz].sid;
            vrec[i].res = (host_pdrec + i)->res;
        }
    }

    // Guardamos resultados
    ofstream ofs("resultados.csv");
    ofs << "tipo,id,e,m,l,ne,nm,nl,ageb_sid,mnz_sid,res" << endl;
    for (auto &r : vrec)
    {
        ofs  << "" << r.stipo << "," << r.id << ","
             << r.e << "," << r.m << ","<<r.l<<","
             << r.ne << "," << r.nm << ","<<r.nl<< ","<<r.sid_ageb<<","<<r.sid_mnz<<","
             << r.res << endl;
    }
    ofs.close();
}