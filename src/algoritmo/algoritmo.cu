#include "hip/hip_runtime.h"
#include "../PuntoPertenciaCUDA.h"
#include "../PuntoPertenecia.h"

extern vector<Recurso> vrec;

extern vector<Estado> vest;
extern vector<Municipio> vmun;
extern vector<Ageb> vageb;
extern vector<Manzana> vmnz;

PDRec host_pdrec = nullptr;
PDRec device_pdrec = nullptr;

PDEP host_dep = nullptr;
PDEP device_dep = nullptr;

PDRefP host_drefp = nullptr;
PDRefP device_drefp = nullptr;

void algoritmo();
void marcaRecursos();

// Aloja memoria
extern void alojaMemRecurso(vector<Recurso> &vrec, PDRec &host_pdrec, PDRec &device_pdrec);
extern void liberaMemRecurso(PDRec &host_pdrec, PDRec &device_pdrec);

extern void alojaMemEstado(vector<Estado> &vest, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);
extern void liberaMemEstado(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);

// kernel
extern __global__ void kernel_polpertenciaEstado(PDEP device_dep, PDRefP device_drefp, PDRec device_pdrec, size_t num_rec, size_t num_pol);

/**
 * @brief
 *
 */
void algoritmo()
{

    size_t tam_rec = vrec.size();

    int canti_hilos = 1024;
    int canti_bloques = (int)ceil((float)tam_rec / canti_hilos) + 1;

    alojaMemRecurso(vrec, host_pdrec, device_pdrec);

    alojaMemEstado(vest, host_dep, device_dep, host_drefp, device_drefp);

    // kernel estado
    kernel_polpertenciaEstado<<<canti_bloques, canti_hilos>>>(device_dep, device_drefp, device_pdrec, tam_rec, vest.size());

    liberaMemEstado(host_dep, device_dep, host_drefp, device_drefp);

    // kernel municipio

    hipMemcpy(host_pdrec, device_pdrec, tam_rec * sizeof(DRec), hipMemcpyDeviceToHost);

    marcaRecursos();

    liberaMemRecurso(host_pdrec, device_pdrec);
}

/**
 * @brief
 *
 */
void marcaRecursos()
{

    size_t tam = vrec.size();

    for (size_t i = 0; i < tam; i++)
    {
        long id_e = (host_pdrec + i)->id_e;
        if (id_e > 0)
        {
            vrec[i].ne = vest[id_e].id;
            vrec[i].res = (host_pdrec + i)->res;
        }
    }

    for (auto &r : vrec)
    {
        cout << "(" << r.stipo << "," << r.id << ") " << r.e << " " << r.ne << " " << r.res << endl;
    }
}