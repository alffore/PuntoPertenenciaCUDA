#include "../PuntoPertenecia.h"
#include "../PuntoPertenciaCUDA.h"

void alojaMemEstado(vector<Estado> &vest, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);
void liberaMemEstado(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp);

/**
 * @brief
 *
 * @param vest
 * @param host_dep
 * @param device_dep
 * @param host_drefp
 * @param device_drefp
 */
void alojaMemEstado(vector<Estado> &vest, PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp)
{
    size_t tam_pol = vest.size();
    size_t tam_coord = 0;

    host_drefp = (PDRefP)malloc(tam_pol * sizeof(DRefP));

    for (auto &e : vest)
    {
        tam_coord += e.vp.size();
    }

    host_dep = (PDEP)malloc(tam_coord * sizeof(DEP));

    size_t pos = 0;
    for (size_t i = 0; i < tam_pol; i++)
    {
        (host_drefp + i)->ini = pos;

        size_t tam_vp = vest[i].vp.size();
        for (size_t j = 0; j < tam_vp; j++)
        {
            (host_dep + pos + j)->x = vest[i].vp[j].x;
            (host_dep + pos + j)->y = vest[i].vp[j].y;
        }

        pos += vest[i].vp.size();
        (host_drefp + i)->fin = pos;
    }

    hipMalloc((void **)&device_dep, tam_coord * sizeof(DEP));
    hipMemcpy(device_dep,host_dep,tam_coord * sizeof(DEP),hipMemcpyHostToDevice);

    hipMalloc((void **)&device_drefp, tam_pol * sizeof(DRefP));
    hipMemcpy(device_drefp,host_drefp,tam_pol * sizeof(DRefP),hipMemcpyHostToDevice);

}

/**
 * @brief
 *
 * @param host_dep
 * @param device_dep
 * @param host_drefp
 * @param device_drefp
 */
void liberaMemEstado(PDEP &host_dep, PDEP &device_dep, PDRefP &host_drefp, PDRefP &device_drefp)
{

    hipFree(device_dep);
    hipFree(device_drefp);

    free(host_dep);
    free(host_drefp);
}
